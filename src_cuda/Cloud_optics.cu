#include "hip/hip_runtime.h"
/*
 * This file is part of a C++ interface to the Radiative Transfer for Energetics (RTE)
 * and Rapid Radiative Transfer Model for GCM applications Parallel (RRTMGP).
 *
 * The original code is found at https://github.com/earth-system-radiation/rte-rrtmgp.
 *
 * Contacts: Robert Pincus and Eli Mlawer
 * email: rrtmgp@aer.com
 *
 * Copyright 2015-2020,  Atmospheric and Environmental Research and
 * Regents of the University of Colorado.  All right reserved.
 *
 * This C++ interface can be downloaded from https://github.com/earth-system-radiation/rte-rrtmgp-cpp
 *
 * Contact: Chiel van Heerwaarden
 * email: chiel.vanheerwaarden@wur.nl
 *
 * Copyright 2020, Wageningen University & Research.
 *
 * Use and duplication is permitted under the terms of the
 * BSD 3-clause license, see http://opensource.org/licenses/BSD-3-Clause
 *
 */

#include <limits>
#include "Cloud_optics.h"

namespace
{
    template<typename TF>__global__
    void compute_from_table_kernel(
            const int ncol, const int nlay, const int nbnd, const Bool* mask,
            const TF* cwp, const TF* re,
            const int nsteps, const TF step_size, const TF offset,
            const TF* tau_table, const TF* ssa_table, const TF* asy_table,
            TF* tau, TF* taussa, TF* taussag)
    {
        const int ibnd = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
        const int icol = blockIdx.z*blockDim.z + threadIdx.z;

        if ( ( icol < ncol) && ( ilay < nlay) && (ibnd < nbnd) )
        {
            const int idx_2d = icol + ilay*ncol;
            const int idx_3d = icol + ilay*ncol + ibnd*nlay*ncol;

            if (mask[idx_2d])
            {
                const int index = min(int((re[idx_2d] - offset) / step_size) + 1, nsteps-1) - 1;
                const int idx_ib = index + ibnd*nsteps;
                const TF fint = (re[idx_2d] - offset) /step_size - (index);
                const TF tau_local = cwp[idx_2d] *
                                     (tau_table[idx_ib] + fint * (tau_table[idx_ib+1] - tau_table[idx_ib]));
                const TF taussa_local = tau_local *
                                     (ssa_table[idx_ib] + fint * (ssa_table[idx_ib+1] - ssa_table[idx_ib]));
                const TF taussag_local = taussa_local *
                                     (asy_table[idx_ib] + fint * (asy_table[idx_ib+1] - asy_table[idx_ib]));

                tau[idx_3d]     = tau_local;
                taussa[idx_3d]  = taussa_local;
                taussag[idx_3d] = taussag_local;
            }
            else
            {
                tau[idx_3d]     = TF(0.);
                taussa[idx_3d]  = TF(0.);
                taussag[idx_3d] = TF(0.);
            }
        }
    }

    template<typename TF>__global__
    void combine_and_store_kernel(const int ncol, const int nlay, const int nbnd, const TF tmin,
                  ATMOS_TYPE* __restrict__ tau,
                  const TF* __restrict__ ltau, const TF* __restrict__ ltaussa,
                  const TF* __restrict__ itau, const TF* __restrict__ itaussa)
    {
        const int ibnd = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
        const int icol = blockIdx.z*blockDim.z + threadIdx.z;

        if ( (icol < ncol) && (ilay < nlay) && (ibnd < nbnd) )
        {
            const int idx = icol + ilay*ncol + ibnd*nlay*ncol;
            const TF tau_t = (ltau[idx] - ltaussa[idx]) + (itau[idx] - itaussa[idx]);

            tau[idx] = ATMOS_TYPE(tau_t);
        }
    }

    template<typename TF>__global__
    void combine_and_store_kernel(const int ncol, const int nlay, const int nbnd, const TF tmin,
                  ATMOS_TYPE* __restrict__ tau, TF* __restrict__ ssa, TF* __restrict__ g,
                  const TF* __restrict__ ltau, const TF* __restrict__ ltaussa, const TF* __restrict__ ltaussag,
                  const TF* __restrict__ itau, const TF* __restrict__ itaussa, const TF* __restrict__ itaussag)
    {
        const int ibnd = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
        const int icol = blockIdx.z*blockDim.z + threadIdx.z;

        if ( (icol < ncol) && (ilay < nlay) && (ibnd < nbnd) )
        {
            const int idx = icol + ilay*ncol + ibnd*nlay*ncol;
            const TF tau_t = ltau[idx] + itau[idx];
            const TF taussa = ltaussa[idx] + itaussa[idx];
            const TF taussag = ltaussag[idx] + itaussag[idx];

            tau[idx] = ATMOS_TYPE(tau_t);
            ssa[idx] = taussa / max(tau_t, tmin);
            g[idx]   = taussag/ max(taussa, tmin);
        }
    }

    template<typename TF>__global__
    void set_mask(const int ncol, const int nlay, const TF min_value,
                  Bool* __restrict__ mask, const TF* __restrict__ values)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilay = blockIdx.y*blockDim.y + threadIdx.y;

        if ( (icol < ncol) && (ilay < nlay) )
        {
            const int idx = icol + ilay*ncol;
            mask[idx] = values[idx] > min_value;
        }
    }
}


Cloud_optics_gpu::Cloud_optics_gpu(
        const Array<Float,2>& band_lims_wvn,
        const Float radliq_lwr, const Float radliq_upr, const Float radliq_fac,
        const Float diamice_lwr, const Float diamice_upr, const Float diamice_fac,
        const Array<Float,2>& lut_extliq, const Array<Float,2>& lut_ssaliq, const Array<Float,2>& lut_asyliq,
        const Array<Float,3>& lut_extice, const Array<Float,3>& lut_ssaice, const Array<Float,3>& lut_asyice) :
    Optical_props_gpu(band_lims_wvn)
{
    const int nsize_liq = lut_extliq.dim(1);
    const int nsize_ice = lut_extice.dim(1);

    this->liq_nsteps = nsize_liq;
    this->ice_nsteps = nsize_ice;
    this->liq_step_size = (radliq_upr - radliq_lwr) / (nsize_liq - Float(1.));
    this->ice_step_size = (diamice_upr - diamice_lwr) / (nsize_ice - Float(1.));

    // Load LUT constants.
    this->radliq_lwr = radliq_lwr;
    this->radliq_upr = radliq_upr;
    this->diamice_lwr = diamice_lwr;
    this->diamice_upr = diamice_upr;

    // Load LUT coefficients.
    this->lut_extliq = lut_extliq;
    this->lut_ssaliq = lut_ssaliq;
    this->lut_asyliq = lut_asyliq;

    // Choose the intermediately rough ice particle category (icergh = 2).
    this->lut_extice.set_dims({lut_extice.dim(1), lut_extice.dim(2)});
    this->lut_ssaice.set_dims({lut_ssaice.dim(1), lut_ssaice.dim(2)});
    this->lut_asyice.set_dims({lut_asyice.dim(1), lut_asyice.dim(2)});

    constexpr int icergh = 2;
    for (int ibnd=1; ibnd<=lut_extice.dim(2); ++ibnd)
        for (int isize=1; isize<=lut_extice.dim(1); ++isize)
        {
            this->lut_extice({isize, ibnd}) = lut_extice({isize, ibnd, icergh});
            this->lut_ssaice({isize, ibnd}) = lut_ssaice({isize, ibnd, icergh});
            this->lut_asyice({isize, ibnd}) = lut_asyice({isize, ibnd, icergh});
        }

    this->lut_extice_gpu = this->lut_extice;
    this->lut_ssaice_gpu = this->lut_ssaice;
    this->lut_asyice_gpu = this->lut_asyice;
    this->lut_extliq_gpu = this->lut_extliq;
    this->lut_ssaliq_gpu = this->lut_ssaliq;
    this->lut_asyliq_gpu = this->lut_asyliq;
}


// Two-stream variant of cloud optics.
void Cloud_optics_gpu::cloud_optics(
        const Array_gpu<Float,2>& clwp, const Array_gpu<Float,2>& ciwp,
        const Array_gpu<Float,2>& reliq, const Array_gpu<Float,2>& deice,
        Optical_props_2str_gpu& optical_props)
{
    const int ncol = clwp.dim(1);
    const int nlay = clwp.dim(2);
    const int nbnd = this->get_nband();

    Optical_props_2str_gpu clouds_liq(ncol, nlay, optical_props);
    Optical_props_2str_gpu clouds_ice(ncol, nlay, optical_props);

    // Set the mask.
    constexpr Float mask_min_value = Float(0.);
    const int block_col_m = 16;
    const int block_lay_m = 16;

    const int grid_col_m  = ncol/block_col_m + (ncol%block_col_m > 0);
    const int grid_lay_m  = nlay/block_lay_m + (nlay%block_lay_m > 0);

    dim3 grid_m_gpu(grid_col_m, grid_lay_m);
    dim3 block_m_gpu(block_col_m, block_lay_m);

    Array_gpu<Bool,2> liqmsk({ncol, nlay});
    set_mask<<<grid_m_gpu, block_m_gpu>>>(
            ncol, nlay, mask_min_value, liqmsk.ptr(), clwp.ptr());

    Array_gpu<Bool,2> icemsk({ncol, nlay});
    set_mask<<<grid_m_gpu, block_m_gpu>>>(
            ncol, nlay, mask_min_value, icemsk.ptr(), ciwp.ptr());

    // Temporary arrays for storage.
    Array_gpu<Float,3> ltau    ({ncol, nlay, nbnd});
    Array_gpu<Float,3> ltaussa ({ncol, nlay, nbnd});
    Array_gpu<Float,3> ltaussag({ncol, nlay, nbnd});

    Array_gpu<Float,3> itau    ({ncol, nlay, nbnd});
    Array_gpu<Float,3> itaussa ({ncol, nlay, nbnd});
    Array_gpu<Float,3> itaussag({ncol, nlay, nbnd});

    const int block_bnd = 14;
    const int block_lay = 1;
    const int block_col = 32;

    const int grid_bnd  = nbnd/block_bnd + (nbnd%block_bnd > 0);
    const int grid_lay  = nlay/block_lay + (nlay%block_lay > 0);
    const int grid_col  = ncol/block_col + (ncol%block_col > 0);

    dim3 grid_gpu(grid_bnd, grid_lay, grid_col);
    dim3 block_gpu(block_bnd, block_lay, block_col);

    // Liquid water
    compute_from_table_kernel<<<grid_gpu, block_gpu>>>(
            ncol, nlay, nbnd, liqmsk.ptr(), clwp.ptr(), reliq.ptr(),
            this->liq_nsteps, this->liq_step_size, this->radliq_lwr,
            this->lut_extliq_gpu.ptr(), this->lut_ssaliq_gpu.ptr(),
            this->lut_asyliq_gpu.ptr(), ltau.ptr(), ltaussa.ptr(), ltaussag.ptr());

    // Ice.
    compute_from_table_kernel<<<grid_gpu, block_gpu>>>(
            ncol, nlay, nbnd, icemsk.ptr(), ciwp.ptr(), deice.ptr(),
            this->ice_nsteps, this->ice_step_size, this->diamice_lwr,
            this->lut_extice_gpu.ptr(), this->lut_ssaice_gpu.ptr(),
            this->lut_asyice_gpu.ptr(), itau.ptr(), itaussa.ptr(), itaussag.ptr());

    constexpr Float eps = std::numeric_limits<Float>::epsilon();

    combine_and_store_kernel<<<grid_gpu, block_gpu>>>(
            ncol, nlay, nbnd, eps,
            optical_props.get_tau().ptr(), optical_props.get_ssa().ptr(), optical_props.get_g().ptr(),
            ltau.ptr(), ltaussa.ptr(), ltaussag.ptr(),
            itau.ptr(), itaussa.ptr(), itaussag.ptr());
}


// 1scl variant of cloud optics.
void Cloud_optics_gpu::cloud_optics(
        const Array_gpu<Float,2>& clwp, const Array_gpu<Float,2>& ciwp,
        const Array_gpu<Float,2>& reliq, const Array_gpu<Float,2>& deice,
        Optical_props_1scl_gpu& optical_props)
{
    const int ncol = clwp.dim(1);
    const int nlay = clwp.dim(2);
    const int nbnd = this->get_nband();

    Optical_props_1scl_gpu clouds_liq(ncol, nlay, optical_props);
    Optical_props_1scl_gpu clouds_ice(ncol, nlay, optical_props);

    // Set the mask.
    constexpr Float mask_min_value = Float(0.);
    const int block_col_m = 16;
    const int block_lay_m = 16;

    const int grid_col_m  = ncol/block_col_m + (ncol%block_col_m > 0);
    const int grid_lay_m  = nlay/block_lay_m + (nlay%block_lay_m > 0);

    dim3 grid_m_gpu(grid_col_m, grid_lay_m);
    dim3 block_m_gpu(block_col_m, block_lay_m);

    Array_gpu<Bool,2> liqmsk({ncol, nlay});
    set_mask<<<grid_m_gpu, block_m_gpu>>>(
            ncol, nlay, mask_min_value, liqmsk.ptr(), clwp.ptr());

    Array_gpu<Bool,2> icemsk({ncol, nlay});
    set_mask<<<grid_m_gpu, block_m_gpu>>>(
            ncol, nlay, mask_min_value, icemsk.ptr(), ciwp.ptr());

    // Temporary arrays for storage.
    Array_gpu<Float,3> ltau    ({ncol, nlay, nbnd});
    Array_gpu<Float,3> ltaussa ({ncol, nlay, nbnd});
    Array_gpu<Float,3> ltaussag({ncol, nlay, nbnd});

    Array_gpu<Float,3> itau    ({ncol, nlay, nbnd});
    Array_gpu<Float,3> itaussa ({ncol, nlay, nbnd});
    Array_gpu<Float,3> itaussag({ncol, nlay, nbnd});

    const int block_bnd = 14;
    const int block_lay = 1;
    const int block_col = 32;

    const int grid_bnd  = nbnd/block_bnd + (nbnd%block_bnd > 0);
    const int grid_lay  = nlay/block_lay + (nlay%block_lay > 0);
    const int grid_col  = ncol/block_col + (ncol%block_col > 0);

    dim3 grid_gpu(grid_bnd, grid_lay, grid_col);
    dim3 block_gpu(block_bnd, block_lay, block_col);

    // Liquid water
    compute_from_table_kernel<<<grid_gpu, block_gpu>>>(
            ncol, nlay, nbnd, liqmsk.ptr(), clwp.ptr(), reliq.ptr(),
            this->liq_nsteps, this->liq_step_size, this->radliq_lwr,
            this->lut_extliq_gpu.ptr(), this->lut_ssaliq_gpu.ptr(),
            this->lut_asyliq_gpu.ptr(), ltau.ptr(), ltaussa.ptr(), ltaussag.ptr());

    // Ice.
    compute_from_table_kernel<<<grid_gpu, block_gpu>>>(
            ncol, nlay, nbnd, icemsk.ptr(), ciwp.ptr(), deice.ptr(),
            this->ice_nsteps, this->ice_step_size, this->diamice_lwr,
            this->lut_extice_gpu.ptr(), this->lut_ssaice_gpu.ptr(),
            this->lut_asyice_gpu.ptr(), itau.ptr(), itaussa.ptr(), itaussag.ptr());

    constexpr Float eps = std::numeric_limits<Float>::epsilon();

    combine_and_store_kernel<<<grid_gpu, block_gpu>>>(
            ncol, nlay, nbnd, eps,
            optical_props.get_tau().ptr(),
            ltau.ptr(), ltaussa.ptr(),
            itau.ptr(), itaussa.ptr());
}
