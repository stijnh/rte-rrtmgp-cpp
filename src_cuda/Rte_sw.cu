#include "hip/hip_runtime.h"
/*
 * This file is part of a C++ interface to the Radiative Transfer for Energetics (RTE)
 * and Rapid Radiative Transfer Model for GCM applications Parallel (RRTMGP).
 *
 * The original code is found at https://github.com/earth-system-radiation/rte-rrtmgp.
 *
 * Contacts: Robert Pincus and Eli Mlawer
 * email: rrtmgp@aer.com
 *
 * Copyright 2015-2020,  Atmospheric and Environmental Research and
 * Regents of the University of Colorado.  All right reserved.
 *
 * This C++ interface can be downloaded from https://github.com/earth-system-radiation/rte-rrtmgp-cpp
 *
 * Contact: Chiel van Heerwaarden
 * email: chiel.vanheerwaarden@wur.nl
 *
 * Copyright 2020, Wageningen University & Research.
 *
 * Use and duplication is permitted under the terms of the
 * BSD 3-clause license, see http://opensource.org/licenses/BSD-3-Clause
 *
 */

#include "Rte_sw.h"
#include "Array.h"
#include "Optical_props.h"

#include "rte_solver_kernels_cuda.h"

namespace
{
    template<typename Float>__global__
    void expand_and_transpose_kernel(
        const int ncol, const int nbnd, const int* __restrict__ limits,
        Float* __restrict__ arr_out, const Float* __restrict__ arr_in)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int ibnd = blockIdx.y*blockDim.y + threadIdx.y;

        if ( ( icol < ncol) && (ibnd < nbnd) )
        {
            const int gpt_start = limits[2*ibnd] - 1;
            const int gpt_end = limits[2*ibnd+1];

            for (int igpt=gpt_start; igpt<gpt_end; ++igpt)
            {
                const int idx_in = ibnd + icol*nbnd;
                const int idx_out = icol + igpt*ncol;

                arr_out[idx_out] = arr_in[idx_in];
            }
        }
    }
}

//namespace rrtmgp_kernel_launcher
//{
//    template<typename Float>
//    void apply_BC(
//            int ncol, int nlay, int ngpt,
//            Bool top_at_1, Array<Float,3>& gpt_flux_dn)
//    {
//        rrtmgp_kernels::apply_BC_0(
//                &ncol, &nlay, &ngpt,
//                &top_at_1, gpt_flux_dn.ptr());
//    }
//
//    template<typename Float>
//    void apply_BC(
//            int ncol, int nlay, int ngpt, Bool top_at_1,
//            const Array<Float,2>& inc_flux, Array<Float,3>& gpt_flux_dn)
//    {
//        rrtmgp_kernels::apply_BC_gpt(
//                &ncol, &nlay, &ngpt, &top_at_1,
//                const_cast<Float*>(inc_flux.ptr()), gpt_flux_dn.ptr());
//    }
//
//    template<typename Float>
//    void apply_BC(
//            int ncol, int nlay, int ngpt, Bool top_at_1,
//            const Array<Float,2>& inc_flux,
//            const Array<Float,1>& factor,
//            Array<Float,3>& gpt_flux)
//    {
//        rrtmgp_kernels::apply_BC_factor(
//                &ncol, &nlay, &ngpt,
//                &top_at_1,
//                const_cast<Float*>(inc_flux.ptr()),
//                const_cast<Float*>(factor.ptr()),
//                gpt_flux.ptr());
//    }
//
//    template<typename Float>
//    void sw_solver_2stream(
//            int ncol, int nlay, int ngpt, Bool top_at_1,
//            const Array<Float,3>& tau,
//            const Array<Float,3>& ssa,
//            const Array<Float,3>& g,
//            const Array<Float,1>& mu0,
//            const Array<Float,2>& sfc_alb_dir_gpt, const Array<Float,2>& sfc_alb_dif_gpt,
//            Array<Float,3>& gpt_flux_up, Array<Float,3>& gpt_flux_dn, Array<Float,3>& gpt_flux_dir)
//    {
//        rrtmgp_kernels::sw_solver_2stream(
//                &ncol, &nlay, &ngpt, &top_at_1,
//                const_cast<Float*>(tau.ptr()),
//                const_cast<Float*>(ssa.ptr()),
//                const_cast<Float*>(g  .ptr()),
//                const_cast<Float*>(mu0.ptr()),
//                const_cast<Float*>(sfc_alb_dir_gpt.ptr()),
//                const_cast<Float*>(sfc_alb_dif_gpt.ptr()),
//                gpt_flux_up.ptr(), gpt_flux_dn.ptr(), gpt_flux_dir.ptr());
//    }


void Rte_sw_gpu::rte_sw(
        const std::unique_ptr<Optical_props_arry_gpu>& optical_props,
        const Bool top_at_1,
        const Array_gpu<Float,1>& mu0,
        const Array_gpu<FLUX_TYPE,2>& inc_flux_dir,
        const Array_gpu<SURFACE_TYPE ,2>& sfc_alb_dir,
        const Array_gpu<SURFACE_TYPE,2>& sfc_alb_dif,
        const Array_gpu<FLUX_TYPE,2>& inc_flux_dif,
        Array_gpu<FLUX_TYPE,3>& gpt_flux_up,
        Array_gpu<FLUX_TYPE,3>& gpt_flux_dn,
        Array_gpu<FLUX_TYPE,3>& gpt_flux_dir)
{
    const int ncol = optical_props->get_ncol();
    const int nlay = optical_props->get_nlay();
    const int ngpt = optical_props->get_ngpt();

    Array_gpu<SURFACE_TYPE,2> sfc_alb_dir_gpt({ncol, ngpt});
    Array_gpu<SURFACE_TYPE,2> sfc_alb_dif_gpt({ncol, ngpt});

    expand_and_transpose(optical_props, sfc_alb_dir, sfc_alb_dir_gpt);
    expand_and_transpose(optical_props, sfc_alb_dif, sfc_alb_dif_gpt);

    const Bool has_dif_bc = false;
    const Bool do_broadband = (gpt_flux_up.dim(3) == 1) ? true : false;

    if (do_broadband)
        throw std::runtime_error("Broadband fluxes not implemented, performance gain on GPU is negligible");
    
    // pass null ptr if size of inc_flux is zero
    const FLUX_TYPE* inc_flux_dif_ptr = (inc_flux_dif.size() == 0) ? nullptr : inc_flux_dif.ptr();

    // Run the radiative transfer solver
    // CvH: only two-stream solutions, I skipped the sw_solver_noscat.
    Rte_solver_kernels_cuda::sw_solver_2stream(
            ncol, nlay, ngpt, top_at_1,
            optical_props->get_tau().ptr(), optical_props->get_ssa().ptr(), optical_props->get_g().ptr(),
            mu0.ptr(),
            sfc_alb_dir_gpt.ptr(), sfc_alb_dif_gpt.ptr(),
            inc_flux_dir.ptr(),
            gpt_flux_up.ptr(), gpt_flux_dn.ptr(), gpt_flux_dir.ptr(),
            has_dif_bc, inc_flux_dif_ptr,
            do_broadband, gpt_flux_up.ptr(), gpt_flux_dn.ptr(), gpt_flux_dir.ptr());

    // CvH: The original fortran code had a call to the reduce here.
    // fluxes->reduce(gpt_flux_up, gpt_flux_dn, gpt_flux_dir, optical_props, top_at_1);
}


void Rte_sw_gpu::expand_and_transpose(
        const std::unique_ptr<Optical_props_arry_gpu>& ops,
        const Array_gpu<SURFACE_TYPE,2> arr_in,
        Array_gpu<SURFACE_TYPE,2>& arr_out)
{
    const int ncol = arr_in.dim(2);
    const int nbnd = ops->get_nband();
    const int block_col = 16;
    const int block_bnd = 14;

    const int grid_col = ncol/block_col + (ncol%block_col > 0);
    const int grid_bnd = nbnd/block_bnd + (nbnd%block_bnd > 0);

    dim3 grid_gpu(grid_col, grid_bnd);
    dim3 block_gpu(block_col, block_bnd);

    Array_gpu<int,2> limits = ops->get_band_lims_gpoint_gpu();

    // Array_gpu<int,2> limits(limitsc);
    expand_and_transpose_kernel<<<grid_gpu, block_gpu>>>(
        ncol, nbnd, limits.ptr(), arr_out.ptr(), arr_in.ptr());
}
